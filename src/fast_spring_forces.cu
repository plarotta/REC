#include "hip/hip_runtime.h"
#include "robot.hpp"



__global__ void compute_spring_forces_kernel(float* masses, float* springs, int total_masses, int total_springs, float t) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < total_springs) {
        // Compute spring forces for spring i
        //TODO: implement kernel
    }
}

void Robot::compute_spring_forces(float t) {
    float* d_masses, *d_springs;
    hipMalloc(&d_masses, total_masses * 15 * sizeof(float));
    hipMalloc(&d_springs, total_springs * 6 * sizeof(float));

    hipMemcpy(d_masses, masses.data(), total_masses * 15 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_springs, springs.data(), total_springs * 6 * sizeof(float), hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int blocks_per_grid = (total_springs + threads_per_block - 1) / threads_per_block;

    compute_spring_forces_kernel<<<blocks_per_grid, threads_per_block>>>(d_masses, d_springs, total_masses, total_springs, t);

    hipMemcpy(masses.data(), d_masses, total_masses * 15 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_masses);
    hipFree(d_springs);
}